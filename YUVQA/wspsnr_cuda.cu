#include "hip/hip_runtime.h"


#include "config.h"

#if MODE==USE_CUDA
#include "wspsnr_cuda.cuh"
#include <chrono>

#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
__global__ void computeWSPSNR(
    const int* ref,         // [NUM_FRAMES * W * H]
    const int* dist,        // [NUM_FRAMES * W * H]
    const double* row_weights, // [H]
    double* sew,                        // [NUM_FRAMES]
    int width,
    int height
) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;

    int x = tx + blockIdx.x * blockDim.x;
    int y = ty + blockIdx.y * blockDim.y;
    int f = blockIdx.z;  // Frame index

    int frame_stride = width * height;
    int idx = f * frame_stride + y * width + x;


    if (x < width && y < height) {
        int ref_val = ref[idx];
        int dist_val = dist[idx];
        int diff = ref_val - dist_val;
       
        int sqdiff = diff * diff;
        double w = __ldg(&row_weights[y]);
        sew[idx] = sqdiff*w;  // Read-only cache optimized
      

    }
}





__global__ void computeWSPSNR_atomic(
    const int* ref,         // [NUM_FRAMES * W * H]
    const int* dist,        // [NUM_FRAMES * W * H]
    const double* row_weights, // [H]
    double* sse_out,                        // [NUM_FRAMES]
    int width,
    int height
) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;

    int x = tx + blockIdx.x * blockDim.x;
    int y = ty + blockIdx.y * blockDim.y;
    int f = blockIdx.z;  // Frame index

    int frame_stride = width * height;
    int idx = f * frame_stride + y * width + x;


    if (x < width && y < height) {
        int ref_val = ref[idx];
        int dist_val = dist[idx];
        int diff = ref_val - dist_val;
        int sse = diff * diff;

        double weight = __ldg(&row_weights[y]);  // Read-only cache optimized
        
        // Atomic add to per-frame MSE output
        atomicAdd(&sse_out[f], (sse * weight));

    }
}


__global__ void computeWSPSNRShared2(
    const int* ref,
    const int* dist,
    const double* row_weights,
    double* partial_sums,   // [NUM_FRAMES][num_blocks]
    int width,
    int height
) {
    __shared__ double smem[NUM_THREADS * NUM_THREADS];  // assuming blockDim.x * blockDim.y <= 256

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;

    int x = tx + blockIdx.x * blockDim.x;
    int y = ty + blockIdx.y * blockDim.y;
    int f = blockIdx.z;

    int frame_stride = width * height;

        double local_sum = 0.0f;
        if (x < width && y < height) {
            int idx = f * frame_stride + y * width + x;
            int diff = ref[idx] - dist[idx];
            int sse = diff * diff;
            double weight = __ldg(&row_weights[y]); // Read-only cache optimized
            local_sum = sse * weight;
        }

        smem[tid] = local_sum;
        __syncthreads();

        // Block-wide reduction
        for (int s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1) {
            if (tid < s)
                smem[tid] += smem[tid + s];
            __syncthreads();
        }


        // Only thread 0 writes the result
        if (tid == 0) {
            int block_idx = blockIdx.y * gridDim.x + blockIdx.x;
            partial_sums[f * gridDim.x * gridDim.y + block_idx] = smem[0];
        }
   
}




__global__ void computeWSPSNRShared(
    const int* ref,
    const int* dist,
    const double* row_weights,
    double* partial_sums,   // [NUM_FRAMES][num_blocks]
    int width,
    int height
) {
    __shared__ double smem[NUM_THREADS * NUM_THREADS];  // assuming blockDim.x * blockDim.y <= 256

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;

    int x = tx + blockIdx.x * blockDim.x;
    int y = ty + blockIdx.y * blockDim.y;
    int f = blockIdx.z;

    int frame_stride = width * height;

        double local_sum = 0.0f;
        //for (int t = 0; t < 10000; t++)
        {
        if (x < width && y < height) {
            int idx = f * frame_stride + y * width + x;
            int diff = ref[idx] - dist[idx];
            int sse = diff * diff;
            double weight = __ldg(&row_weights[y]); // Read-only cache optimized
            local_sum = sse * weight;
        }
    }
        smem[tid] = local_sum;
        __syncthreads();

        // Block-wide reduction
        for (int s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1) {
            if (tid < s)
                smem[tid] += smem[tid + s];
            __syncthreads();
        }


        // Only thread 0 writes the result
        if (tid == 0) {
            int block_idx = blockIdx.y * gridDim.x + blockIdx.x;
            partial_sums[f * gridDim.x * gridDim.y + block_idx] = smem[0];
        }
    
}

hipError_t run_process_cuda_atomic(const int* ref, const int* rec, const double* weights, double* wspsnr_frame, const double w_sum, const  int w, int h, int bitDepth, int nf = 1)
{

    int* dev_ref = 0;
    int* dev_rec = 0;
    //__constant__  double* dev_weights = new double[h];
    double* dev_weights = 0;
    double* dev_sseout = 0;
    hipError_t cudaStatus;
    double MAX_VALUE = (255.0 * (1 << (bitDepth - 8)));



    double* host_sseout = new double[nf];
    int size = w * h * nf;

    /*sdouble* h_row_weights = new double[h];
    for (int y = 0; y < h; ++y)
    {
        double theta = ((y + 0.5f) / h - 0.5f) * M_PI;
        h_row_weights[y] = cosf(theta);
    }*/
    //hipMemcpyToSymbol(HIP_SYMBOL(dev_weights), weights, size * sizeof(double));

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_ref, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rec, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_sseout, nf * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_weights, h * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_ref, ref, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rec, rec, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_weights, weights, h * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

   
    dim3 blockDim(NUM_THREADS, NUM_THREADS);
    dim3 gridDim((w + NUM_THREADS - 1) / NUM_THREADS, (h + NUM_THREADS - 1) / NUM_THREADS, nf);

    computeWSPSNR_atomic << < gridDim, blockDim >> > (dev_ref, dev_rec, dev_weights, dev_sseout, w, h);


    cudaStatus = hipMemcpy(host_sseout, dev_sseout, nf * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    for (int f = 0; f < nf; f++)
        wspsnr_frame[f] = 10 * std::log10((MAX_VALUE * MAX_VALUE * w_sum) / host_sseout[f]);


Error:
    hipFree(dev_ref);
    hipFree(dev_rec);
    hipFree(dev_weights);
    hipFree(dev_sseout);
    hipFree(host_sseout);

    return cudaStatus;
}


hipError_t run_process_cuda_shared_mem(const int* ref, const int* rec, const double* weights, double* wspsnr_frame, const double w_sum, const  int w, int h, int bitDepth, int nf = 1)
{

    int* dev_ref = 0;
    int* dev_rec = 0;
    //__constant__  double* dev_weights = new double[h];
    double* dev_weights = 0;
    double* dev_partial_sum = 0;
    hipError_t cudaStatus;
    double MAX_VALUE = (255.0 * (1 << (bitDepth - 8)));

    int size = w * h * nf;

    dim3 blockDim(NUM_THREADS, NUM_THREADS);
    dim3 gridDim((w + NUM_THREADS - 1) / NUM_THREADS, (h + NUM_THREADS - 1) / NUM_THREADS, nf);
    
    int num_blocks = gridDim.x * gridDim.y;
    int partial_sum_num = num_blocks * gridDim.z;   //num_frames * num_blocks
    double* h_partial_sum = new double[partial_sum_num];

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_ref, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rec, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_partial_sum, partial_sum_num * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_weights, h * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_ref, ref, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rec, rec, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_weights, weights, h * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    auto st = std::chrono::high_resolution_clock::now();
    computeWSPSNRShared << <gridDim, blockDim >> > ( dev_ref, dev_rec, dev_weights, dev_partial_sum,w, h );

    cudaStatus = hipMemcpy(h_partial_sum, dev_partial_sum, partial_sum_num * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    for (int f = 0; f < nf; ++f) {
        double weighted_mse = 0.0f;
        for (int b = 0; b < num_blocks; ++b) {
            weighted_mse += h_partial_sum[f * num_blocks + b];

            wspsnr_frame[f] = 10 * std::log10((MAX_VALUE * MAX_VALUE * w_sum) / weighted_mse);
        }
    }
    auto et = std::chrono::high_resolution_clock::now();
    double duration = std::chrono::duration_cast<std::chrono::milliseconds>(et - st).count() / 1000.0;
    std::cout << duration << std::endl;


Error:
    hipFree(dev_ref);
    hipFree(dev_rec);
    hipFree(dev_weights);
    free(h_partial_sum);
    hipFree(dev_partial_sum);

    return cudaStatus;
}


void intit_device_buffer(int** dev_ref, int** dev_rec, double** dev_weights , double** dev_partial_sum, const  int w, int h, const double* weights,int nf=1)
{
    
    hipError_t cudaStatus;

    int size = w * h * nf;

    dim3 blockDim(NUM_THREADS, NUM_THREADS);
    dim3 gridDim((w + NUM_THREADS - 1) / NUM_THREADS, (h + NUM_THREADS - 1) / NUM_THREADS, nf);

    int num_blocks = gridDim.x * gridDim.y;
    int partial_sum_num = num_blocks * gridDim.z;   //num_frames * num_blocks

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&(*dev_ref), size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        
    }

    cudaStatus = hipMalloc((void**)&(*dev_rec), size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
       
    }

    cudaStatus = hipMalloc((void**)&(*dev_partial_sum), size * sizeof(double));
    //cudaStatus = hipMalloc((void**)&(*dev_partial_sum), partial_sum_num * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
       
    }

    cudaStatus = hipMalloc((void**)&(*dev_weights), h * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        
    }


    cudaStatus = hipMemcpy((*dev_weights), weights, h * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
      
    }
}

hipError_t run_process_cuda_shared_mem(const int* ref, const int* rec, int* dev_ref, int* dev_rec, double* dev_weights, double* dev_partial_sum,const double* weights,  double* wspsnr_frame, const double w_sum, const  int w, int h, int bitDepth, int nf = 1)
{

    /*int* dev_ref = 0;
    int* dev_rec = 0;
    //__constant__  double* dev_weights = new double[h];
    double* dev_weights = 0;
    double* dev_partial_sum = 0;*/

    hipError_t cudaStatus;
    double MAX_VALUE = (255.0 * (1 << (bitDepth - 8)));

    int size = w * h * nf;

    dim3 blockDim(NUM_THREADS, NUM_THREADS);
    dim3 gridDim((w + NUM_THREADS - 1) / NUM_THREADS, (h + NUM_THREADS - 1) / NUM_THREADS, nf);

    int num_blocks = gridDim.x * gridDim.y;
    int partial_sum_num = num_blocks * gridDim.z;   //num_frames * num_blocks
    
    double* h_partial_sum = new double[partial_sum_num];

    // Allocate GPU buffers for three vectors (two input, one output)    .
    /*cudaStatus = hipMalloc((void**)&dev_ref, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rec, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_partial_sum, partial_sum_num * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_weights, h * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }*/

   
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_ref, ref, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rec, rec, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    /*
    cudaStatus = hipMemcpy(dev_weights, weights, h * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }*/
    auto st = std::chrono::high_resolution_clock::now();
    computeWSPSNRShared << <gridDim, blockDim >> > (dev_ref, dev_rec, dev_weights, dev_partial_sum, w, h);

    cudaStatus = hipMemcpy(h_partial_sum, dev_partial_sum, partial_sum_num * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    for (int f = 0; f < nf; ++f) {
        double weighted_mse = 0.0f;
        for (int b = 0; b < num_blocks; ++b) {
            weighted_mse += h_partial_sum[f * num_blocks + b];

            wspsnr_frame[f] = 10 * std::log10((MAX_VALUE * MAX_VALUE * w_sum) / weighted_mse);
        }
    }
    auto et = std::chrono::high_resolution_clock::now();
    double duration = std::chrono::duration_cast<std::chrono::milliseconds>(et - st).count() / 1000.0;
    std::cout << duration << std::endl;


Error:
    //hipFree(dev_ref);
    //hipFree(dev_rec);
    //hipFree(dev_weights);
    //hipFree(dev_partial_sum);
    free(h_partial_sum);
    
    return cudaStatus;
}


hipError_t run_process_cuda_shared_mem_2streams(const int* ref, const int* rec, const double* weights, double* wspsnr_frame, const double w_sum, const  int w, int h, int bitDepth, int nf = 1)
{

    hipError_t cudaStatus;
    double MAX_VALUE = (255.0 * (1 << (bitDepth - 8)));
    int size = w * h ;
    int framesize = (w * h);

    dim3 blockDim(NUM_THREADS, NUM_THREADS);
    dim3 gridDim((w + NUM_THREADS - 1) / NUM_THREADS, (h + NUM_THREADS - 1) / NUM_THREADS, nf/2);

    int num_blocks = gridDim.x * gridDim.y;
    int partial_sum_num = num_blocks * gridDim.z;   //num_frames * num_blocks

    double* h_partial_sum = new double[partial_sum_num* nf];


    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    double* dev_weights = 0;

    cudaStatus = hipMalloc((void**)&dev_weights, h * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMemcpy(dev_weights, weights, h * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    int* dev_ref = 0;
    int* dev_rec = 0;
    double* dev_partial_sum = 0;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_ref, framesize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rec, framesize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_partial_sum, num_blocks * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_ref, ref, framesize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_rec, rec, framesize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    computeWSPSNRShared << <gridDim, blockDim ,0, stream1 >> > (dev_ref, dev_rec, dev_weights, dev_partial_sum, w, h);


    int* dev_ref1 = 0;
    int* dev_rec1 = 0;
    double* dev_partial_sum1 = 0;
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_ref1, framesize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rec1, framesize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_partial_sum1, num_blocks * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_ref1, ref+ framesize, framesize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_rec1, rec+ framesize, framesize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    computeWSPSNRShared << <gridDim, blockDim, 0, stream2 >> > (dev_ref1, dev_rec1, dev_weights, dev_partial_sum1, w, h);


    // Optional: wait for both to finish
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    cudaStatus = hipMemcpy(h_partial_sum, dev_partial_sum, num_blocks * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(h_partial_sum+ num_blocks, dev_partial_sum1, num_blocks * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    for (int f = 0; f < nf; ++f) {
        double weighted_mse = 0.0f;
        for (int b = 0; b < num_blocks; ++b) {
            weighted_mse += h_partial_sum[f * num_blocks + b];

            wspsnr_frame[f] = 10 * std::log10((MAX_VALUE * MAX_VALUE * w_sum) / weighted_mse);
        }
    }
    // Cleanup
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);


Error:
    hipFree(dev_ref);
    hipFree(dev_rec);
    hipFree(dev_ref1);
    hipFree(dev_rec1);
    hipFree(dev_weights);
    free(h_partial_sum);


    return cudaStatus;
}



hipError_t run_process_cuda(const int* ref, const int* rec, int* dev_ref, int* dev_rec, double* dev_weights, double* sqdiff_weighted, const double* weights, double* wspsnr_frame, const double w_sum, const  int w, int h, int bitDepth, int nf = 1)
{


    hipError_t cudaStatus;
    double MAX_VALUE = (255.0 * (1 << (bitDepth - 8)));

    int size = w * h * nf;

    dim3 blockDim(NUM_THREADS, NUM_THREADS);
    dim3 gridDim((w + NUM_THREADS - 1) / NUM_THREADS, (h + NUM_THREADS - 1) / NUM_THREADS, nf);

    int num_blocks = gridDim.x * gridDim.y;
    //int partial_sum_num = num_blocks * gridDim.z;   //num_frames * num_blocks

    //double* h_partial_sum = new double[partial_sum_num];

    // Allocate GPU buffers for three vectors (two input, one output)    .
    /*cudaStatus = hipMalloc((void**)&dev_ref, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rec, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_partial_sum, partial_sum_num * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_weights, h * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }*/

    
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_ref, ref, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rec, rec, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    /*
    cudaStatus = hipMemcpy(dev_weights, weights, h * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }*/
    auto st = std::chrono::high_resolution_clock::now();
    computeWSPSNR << <gridDim, blockDim >> > (dev_ref, dev_rec, dev_weights, sqdiff_weighted, w, h);

   /* cudaStatus = hipMemcpy(h_partial_sum, sqdiff_weighted, partial_sum_num * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }*/


    for (int f = 0; f < nf; ++f) {

        double weighted_sse = thrust::reduce(
            thrust::device_pointer_cast(sqdiff_weighted + (w * h * f)),
            thrust::device_pointer_cast(sqdiff_weighted + (w * h * (f + 1))), 
             0.0f, thrust::plus<double>());

        wspsnr_frame[f] = 10 * std::log10((MAX_VALUE * MAX_VALUE * w_sum) / weighted_sse);
    }
    auto et = std::chrono::high_resolution_clock::now();
    double duration = std::chrono::duration_cast<std::chrono::milliseconds>(et - st).count() / 1000.0;
    std::cout << duration << std::endl;


Error:
    //hipFree(dev_ref);
    //hipFree(dev_rec);
    //hipFree(dev_weights);
    //hipFree(dev_partial_sum);
    //free(h_partial_sum);

    return cudaStatus;
}







#endif
